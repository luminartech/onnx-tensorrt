#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2018, NVIDIA CORPORATION. All rights reserved.
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.  IN NO EVENT SHALL
 * THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */

#include "BinaryOp.hpp"
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <cassert>

#include <thrust/transform.h>
#include <thrust/system/cuda/execution_policy.h>

#include "CuOpHelpers.hpp"

#define CAPTURE(...) [__VA_ARGS__]
#define BINARY_TRANSFORM(captured_type, func) \
  thrust::transform(thrust::cuda::par.on(stream), \
                    idata0, idata0 + num_elements, idata1, odata, \
                    captured_type __device__ (float x, float y) { return func; })


template <typename Data>
int BinaryOpPlugin::doEnqueue(int batchSize,
                              const void *const *inputs, void **outputs,
                              void *workspace, hipStream_t stream) {
  size_t num_elements = batchSize * get_num_elements(this->getInputDims(0));

  Data const* idata0 = static_cast<Data const*>(inputs[0]);
  Data const* idata1 = static_cast<Data const*>(inputs[1]);
  Data*       odata  = static_cast<Data*      >(outputs[0]);

  // Note: These local-scope copies are needed for lambda capture
  float alpha = _alpha;
  float beta = _beta;

  switch( _op_type ) {
    case LESS:  BINARY_TRANSFORM(CAPTURE(alpha), ((x<y) ? 0.0f : 1.0f)); break;
    default: return -1;
  }
  return hipGetLastError() != hipSuccess;
}

int BinaryOpPlugin::enqueue(int batchSize,
                            const void *const *inputs, void **outputs,
                            void *workspace, hipStream_t stream) {
  if (getDataType()==nvinfer1::DataType::kFLOAT) {        
    return doEnqueue<float>(batchSize, inputs, outputs, workspace, stream);
  } else {
#if CUDART_VERSION < 9000
    throw std::runtime_error("FP16 plugin is not supported for CUDA < 9.0");
#else    
    return doEnqueue<__half>(batchSize, inputs, outputs, workspace, stream);
#endif  
  }
}

bool BinaryOpPlugin::supportsFormat(nvinfer1::DataType type,
                                    nvinfer1::PluginFormat format) const {
  return (type == nvinfer1::DataType::kFLOAT || type == nvinfer1::DataType::kHALF);
}
